
#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 3
#define ROWS 2
__global__ void matadd(int *a, int *b, int *c)
{
 int x = blockIdx.x;
 int y = blockIdx.y;
 int i = (COLUMNS*y) + x;
 c[i] = a[i] + b[i];
}

/* ------------- COMPUTATION DONE ON GPU ----------------------------*/

int main()
{
 int a[ROWS][COLUMNS], b[ROWS][COLUMNS], c[ROWS][COLUMNS];
 int *dev_a, *dev_b, *dev_c;
 hipMalloc((void **) &dev_a, ROWS*COLUMNS*sizeof(int));
 hipMalloc((void **) &dev_b, ROWS*COLUMNS*sizeof(int));
 hipMalloc((void **) &dev_c, ROWS*COLUMNS*sizeof(int));
 for (int i = 0; i < ROWS; i++) // Fill Arrays
 for (int j = 0; j < COLUMNS; j++)
 {
 a[i][j] = 1;
 b[i][j] = 3;
 }
 hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
 dim3 grid(COLUMNS,ROWS);
 matadd<<<grid,1>>>(dev_a, dev_b, dev_c);
 hipMemcpy(c, dev_c, ROWS*COLUMNS*sizeof(int),hipMemcpyDeviceToHost);

/* ------------- COMPUTATION DONE ON HOST CPU ---------------------------*/

 for (int i = 0; i < ROWS; i++) // Output Arrays
 {
 for (int j = 0; j < COLUMNS; j++)
 {
 printf("[%d][%d]=%d ",i,j,c[i][j]);
 }
 printf("\n");
 }
 return 0;
}
