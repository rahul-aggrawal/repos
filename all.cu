#include "hip/hip_runtime.h"
//HELLO.CU
#include "stdio.h"
int main()
{
 printf("Hello, world\n");
 return 0;
}



//ADD.CU



#include <stdio.h>
#include <stdlib.h>
__global__ 
void add(int *a, int *b, int *c) 
{
*c = *a + *b;
}
int main() 
{
  int a, b, c;
  // host copies of variables a, b & c
  int *d_a, *d_b, *d_c;
  // device copies of variables a, b & c
  int size = sizeof(int);
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Setup input values  
  c = 0;
  a = 3;
  b = 6;
  // Copy inputs to device
  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU
  add<<<1,1>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipError_t err = hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
  if(err!=hipSuccess) 
  {
      printf("CUDA error copying to Host: %s\n", hipGetErrorString(err));
  }
  printf("result is %d\n",c);
  // Cleanup
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
  return 0;
}



//VECADD.CU



#include "stdio.h"
#define N 10
__global__ void add(int *a, int *b, int *c)
{
 int tID = blockIdx.x;
 if (tID < N)
 {
 c[tID] = a[tID] + b[tID];
 }
}
int main()
{
 int a[N], b[N], c[N];
 int *dev_a, *dev_b, *dev_c;
 hipMalloc((void **) &dev_a, N*sizeof(int));
 hipMalloc((void **) &dev_b, N*sizeof(int));
 hipMalloc((void **) &dev_c, N*sizeof(int));
 // Fill Arrays
 for (int i = 0; i < N; i++)
 {
 a[i] = i,
 b[i] = 1;
 }
 hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, N*sizeof(int), hipMemcpyHostToDevice);
 add<<<N,1>>>(dev_a, dev_b, dev_c);
 hipMemcpy(c, dev_c, N*sizeof(int), hipMemcpyDeviceToHost);
 for (int i = 0; i < N; i++)
 {
 printf("%d + %d = %d\n", a[i], b[i], c[i]);
 }
 return 0;
}



//MATADD.CU



#include "stdio.h"
#define COLUMNS 3
#define ROWS 2
__global__ void matadd(int *a, int *b, int *c)
{
 int x = blockIdx.x;
 int y = blockIdx.y;
 int i = (COLUMNS*y) + x;
 c[i] = a[i] + b[i];
}

/* ------------- COMPUTATION DONE ON GPU ----------------------------*/

int main()
{
 int a[ROWS][COLUMNS], b[ROWS][COLUMNS], c[ROWS][COLUMNS];
 int *dev_a, *dev_b, *dev_c;
 hipMalloc((void **) &dev_a, ROWS*COLUMNS*sizeof(int));
 hipMalloc((void **) &dev_b, ROWS*COLUMNS*sizeof(int));
 hipMalloc((void **) &dev_c, ROWS*COLUMNS*sizeof(int));
 for (int i = 0; i < ROWS; i++) // Fill Arrays
 for (int j = 0; j < COLUMNS; j++)
 {
 a[i][j] = 1;
 b[i][j] = 3;
 }
 hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
 dim3 grid(COLUMNS,ROWS);
 matadd<<<grid,1>>>(dev_a, dev_b, dev_c);
 hipMemcpy(c, dev_c, ROWS*COLUMNS*sizeof(int),hipMemcpyDeviceToHost);

/* ------------- COMPUTATION DONE ON HOST CPU ---------------------------*/

 for (int i = 0; i < ROWS; i++) // Output Arrays
 {
 for (int j = 0; j < COLUMNS; j++)
 {
 printf("[%d][%d]=%d ",i,j,c[i][j]);
 }
 printf("\n");
 }
 return 0;
}




//MATMUL.CU



#include "stdio.h"
#define COLUMNS 3
#define ROWS 3
__global__ void matmul(int *a, int *b, int *c, int width)
{
  int row = blockIdx.y*width+threadIdx.y;
  int col = blockIdx.x*width+threadIdx.x;
  if(row<width && col <width) 
  {
    int product_val = 0;
    for(int k=0;k<width;k++) 
    {
      product_val += a[row*width+k]*b[k*width+col];
    }
    c[row*width+col] = product_val;
  }
}

/* ------------- COMPUTATION DONE ON GPU ----------------------------*/

int main()
{
 int a[ROWS][COLUMNS], b[ROWS][COLUMNS], c[ROWS][COLUMNS];
 int *dev_a, *dev_b, *dev_c;
 int N = 3;
 hipMalloc((void **) &dev_a, ROWS*COLUMNS*sizeof(int));
 hipMalloc((void **) &dev_b, ROWS*COLUMNS*sizeof(int));
 hipMalloc((void **) &dev_c, ROWS*COLUMNS*sizeof(int));
 for (int i = 0; i < ROWS; i++) // Fill Arrays
 for (int j = 0; j < COLUMNS; j++)
 {
 a[i][j] = 1;
 b[i][j] = 3;
 }
 hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
 dim3 grid(COLUMNS,ROWS);
 matmul<<<grid,1>>>(dev_a, dev_b, dev_c,N);
 hipMemcpy(c, dev_c, ROWS*COLUMNS*sizeof(int),hipMemcpyDeviceToHost);

/* ------------- COMPUTATION DONE ON HOST CPU ---------------------------*/

 for (int i = 0; i < ROWS; i++) // Output Arrays
 {
 for (int j = 0; j < COLUMNS; j++)
 {
 printf("[%d][%d]=%d ",i,j,c[i][j]);
 }
 printf("\n");
 }
 return 0;
}
