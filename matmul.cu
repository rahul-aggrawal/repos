
#include <hip/hip_runtime.h>
#include "stdio.h"
#define COLUMNS 3
#define ROWS 3
__global__ void matmul(int *a, int *b, int *c, int width)
{
  int row = blockIdx.y*width+threadIdx.y;
  int col = blockIdx.x*width+threadIdx.x;
  if(row<width && col <width) 
  {
    int product_val = 0;
    for(int k=0;k<width;k++) 
    {
      product_val += a[row*width+k]*b[k*width+col];
    }
    c[row*width+col] = product_val;
  }
}

/* ------------- COMPUTATION DONE ON GPU ----------------------------*/

int main()
{
 int a[ROWS][COLUMNS], b[ROWS][COLUMNS], c[ROWS][COLUMNS];
 int *dev_a, *dev_b, *dev_c;
 int N = 3;
 hipMalloc((void **) &dev_a, ROWS*COLUMNS*sizeof(int));
 hipMalloc((void **) &dev_b, ROWS*COLUMNS*sizeof(int));
 hipMalloc((void **) &dev_c, ROWS*COLUMNS*sizeof(int));
 for (int i = 0; i < ROWS; i++) // Fill Arrays
 for (int j = 0; j < COLUMNS; j++)
 {
 a[i][j] = 1;
 b[i][j] = 3;
 }
 hipMemcpy(dev_a, a, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
 hipMemcpy(dev_b, b, ROWS*COLUMNS*sizeof(int),hipMemcpyHostToDevice);
 dim3 grid(COLUMNS,ROWS);
 matmul<<<grid,1>>>(dev_a, dev_b, dev_c,N);
 hipMemcpy(c, dev_c, ROWS*COLUMNS*sizeof(int),hipMemcpyDeviceToHost);

/* ------------- COMPUTATION DONE ON HOST CPU ---------------------------*/

 for (int i = 0; i < ROWS; i++) // Output Arrays
 {
 for (int j = 0; j < COLUMNS; j++)
 {
 printf("[%d][%d]=%d ",i,j,c[i][j]);
 }
 printf("\n");
 }
 return 0;
}
